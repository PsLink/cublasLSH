#include "hip/hip_runtime.h"
#include "header.h"
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hipblas.h>
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

//Print matrix A(rows_A, cols_A) storage in column-major format
void print_matrix(const double *A, int rows_A, int cols_A) {
    for (int i = 0; i < rows_A; ++i) {
        for (int j = 0; j < cols_A; ++j) {
            std::cout << A[j * rows_A + i] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

// Multiply the arrays A and B on GPU and save the result in C
// C(m,n) = A(m,k) * B(k,n)
void gpu_blas_mmul(hipblasHandle_t &handle, const double *A, const double *B, double *C, const int m, const int k, const int n) {
    //ld = leading dimension
    int lda = m, ldb = k, ldc = m;
    const double alf = 1;
    const double bet = 0;
    const double *alpha = &alf;
    const double *beta = &bet;

    // Do the actual multiplication
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}


void excuteLSH(int numOfPoint, int numOfHash, int radius,
               vector<vector<int> > &Bucket,
               vector<vector<int> > &DataSet,
               vector<vector<vector<int> > > &hashTable,
               int &MAXHASHVALUE) {

    int dim = DataSet[0].size(), batch = 1000;
    srand((unsigned) time(NULL));

    double *hashFunctions, *b;
    hashFunctions = (double *)calloc(numOfHash * dim, sizeof(double));
    b = (double *)calloc(numOfHash, sizeof(double));

    // build lsh, line by line
    for ( int i = 0; i < numOfHash; i++ )
    {
        // generate a hash function
        for (int j = 0; j < dim; j++) {
            hashFunctions[IDX2C(j, i, dim)] = genCauchyRandom() / radius;
        }

        b[i] = genUniformRandom(0, radius) / radius;
    }

    //print_matrix(hashFunctions,dim,numOfHash);

    double *d_hashFunctions, *d_buckeID, *bucketID, *data, *d_data;

    data = (double *)calloc(dim * batch, sizeof(double));
    bucketID = (double *)calloc(numOfHash * batch, sizeof(double));

    hipMalloc((void**)&d_buckeID, batch * numOfHash * sizeof(double));
    hipMalloc((void**)&d_data, batch * dim * sizeof(double));
    hipMalloc((void**)&d_hashFunctions, numOfHash * dim * sizeof(double));

    hipMemcpy(d_hashFunctions, hashFunctions, numOfHash * dim * sizeof(double), hipMemcpyDefault);


    vector<vector<int> > allbucketID;
    vector<int> minID;

    allbucketID.resize(numOfPoint);
    for (int i = 0; i < numOfPoint; i++)
        allbucketID[i].resize(numOfHash);

    minID.resize(numOfHash);

    for (int i = 0; i < numOfHash; i++)
        minID[i] = 0;

    int tmp;
    printf("Start LSH\n");

    // Create a handle for CUBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    for (int i = 0; i < numOfPoint; i += batch) {

        for (int k = 0; k < batch; k++)
            for (int j = 0; j < dim; j++)
                data[IDX2C(k, j, batch)] = DataSet[k + i][j];

        hipMemcpy(d_data, data, dim * batch * sizeof(double), hipMemcpyDefault);

        gpu_blas_mmul(handle, d_data, d_hashFunctions, d_buckeID, batch, dim, numOfHash);

        hipMemcpy(bucketID, d_buckeID, numOfHash * batch * sizeof(double), hipMemcpyDeviceToHost);

        for (int k = 0; k < batch; k++) {
            for (int j = 0; j < numOfHash; j++) {
                //tmp = floor(bucketID[k * numOfHash + j]);
                tmp = floor(bucketID[IDX2C(k, j, batch)] + b[j]);

                allbucketID[i + k][j] = tmp;
                if (tmp < minID[j])
                    minID[j] = tmp;
            }
        }
    }

    // Destroy the handle
    hipblasDestroy(handle);

    hipFree(d_hashFunctions);
    hipFree(d_data);
    hipFree(d_buckeID);
    free(data);
    free(bucketID);
    free(hashFunctions);
    free(b);

    //freopen("lsh_result.txt", "w", stdout);
    int actualID;
    printf("finish LSH\n");


    for (int i = 0; i < numOfPoint; i++) {
        for (int j = 0; j < numOfHash; j++) {
            actualID = allbucketID[i][j] - minID[j];
            if (actualID > hashTable[j].size()) {
                hashTable[j].resize(actualID + 500);
            }
            hashTable[j][actualID].push_back(i);
        }
    }
}